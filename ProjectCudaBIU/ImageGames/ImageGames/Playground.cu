#include "hip/hip_runtime.h"

#include "PlayGround.cuh"

//#include"opencv2/opencv.hpp"
//#include"opencv2/gpu/gpu.hpp"

__global__ void DoGradientWork(const uchar* d_inputArray, uchar* d_outputArray,int numRows,int step,int numCols )
{
	int idX = threadIdx.x;
	int idY = threadIdx.y;
	int gdX = 0;
	int gdY = 0;
	//asuming 256 threads per block 
	for (int i = 0; i < numCols /256; i++)
	{
		if (blockIdx.x != 0)
		{
			if (idX != 0)
			{
				gdX += d_inputArray[(blockIdx.x - 1)*step + i * 256 - 1];
				gdY+= d_inputArray[(blockIdx.x - 1)*step + i * 256 - 1];
			}
			if (idX*i < numCols - 1)
			{
				gdX -= d_inputArray[(blockIdx.x - 1)*step + i * 256 + 1];
				gdY += d_inputArray[(blockIdx.x - 1)*step + i * 256 + 1];
			}
			gdY+= 2* d_inputArray[(blockIdx.x - 1)*step + i * 256 ];
		}
		if (blockIdx.x != numRows - 1)
		{
			if (idX != 0)
			{
				gdX += d_inputArray[(blockIdx.x + 1)*step + i * 256 - 1];
				gdY -= d_inputArray[(blockIdx.x + 1)*step + i * 256 - 1];
			}
			if (idX*i < numCols - 1)
			{
				gdX -= d_inputArray[(blockIdx.x + 1)*step + i * 256 + 1];
				gdY -= d_inputArray[(blockIdx.x + 1)*step + i * 256 + 1];
			}
			gdY -= 2 * d_inputArray[(blockIdx.x + 1)*step + i * 256];

		}
		if (idX != 0)
		{
			gdX += 2*d_inputArray[(blockIdx.x )*step + i * 256 - 1];
		}
		if (idX*i < numCols - 1)
		{
			gdX-=2* d_inputArray[(blockIdx.x )*step + i * 256 + 1];
		}
		
		d_outputArray[blockIdx.x*step + i * 256] = floorSqrt(gdX*gdX + gdY * gdY);
	}
}

__device__ int floorSqrt(int x)
{
	// Base cases 
	if (x == 0 || x == 1)
		return x;

	// Do Binary Search for floor(sqrt(x)) 
	int start = 1, end = x, ans;
	while (start <= end)
	{
		int mid = (start + end) / 2;

		// If x is a perfect square 
		if (mid*mid == x)
			return mid;

		// Since we need floor, we update answer when mid*mid is  
		// smaller than x, and move closer to sqrt(x) 
		if (mid*mid < x)
		{
			start = mid + 1;
			ans = mid;
		}
		else // If mid*mid is greater than x 
			end = mid - 1;
	}
	return ans;
}
